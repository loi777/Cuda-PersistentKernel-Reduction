#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "chrono.c"
#include "log.c"

#define NP 28            // Number of processors
#define BLOCKS 2         // Number of blocks per processor
#define THREADS 1024     // Number of threads per block
#define CHECK(A, M, ...) \
  check((A), __FILE__, __LINE__, __func__, (M), ##__VA_ARGS__)

typedef unsigned int u_int;

#define MAX(a,b) ((a)>(b) ? (a) : (b))

//=========================================================================


// funcao atomica para Max de float retirada diretamente da internet
__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}



//---------------------



__global__ void reduceMax_persist(float *max, float *input, int nElements) {
  u_int blockSize = THREADS*2;                      // the size of the vector segment to reduce
  u_int blockStartPosi = blockIdx.x * blockSize;    // the starting index of current block
  u_int startIndexAdd = blockSize * NP*BLOCKS;         // every loop adds to block start position

  u_int threadsActive;                              // how many threads the block is using

  u_int indexToCompare;                             // used to know where to compare
  u_int currentI;                                   // this saves a few calculations each loop

  // Initial loop where we scan MOST of the vector
  // there will be 1 value to compare at the index where every block started

  for (; blockStartPosi < nElements; blockStartPosi += startIndexAdd) {
    // OUTER LOOP, where we increment the position of every block

    indexToCompare = 1;
    currentI = startIndexAdd + (threadIdx.x * 2);

    // INSIDE LOOP, where threads compare values inside block
    for (threadsActive = THREADS; threadsActive > 0; threadsActive /= 2) {
      if (threadsActive < threadIdx.x) {
        continue;     // skip this thread, for it is inactive
      }

      if (currentI > nElements) {
        continue;     // skip, because this thread is outside the current array
      }

      input[currentI] = MAX(input[currentI] , input[currentI + (indexToCompare)]);

      indexToCompare *= 2;
      currentI *= 2;
    }

    // Exiting the inside loop there will be missing 1 last process
    // let thread 0 do this last comparison
    if (threadIdx.x == 0) {
      input[currentI] = MAX(input[currentI] , input[currentI + (indexToCompare)]);
    }
  }

  // Final comparison utilizing ATOMIC operations
  // We compare the "winner" of every block against the other

  if (threadIdx.x == 0) {   // only first thread does this final comparison
    
  }

}



__global__ void reduceMax_atomic_persist(float *max, float *input, int nElements) {
  u_int i;
  #define INITIAL (blockDim.x * blockIdx.x + threadIdx.x)
  #define NTA (gridDim.x * blockDim.x)

  for (i=INITIAL; i<nElements ;i+=NTA)
    if (input[i] > *max)
      *max = input[i];
}



//-------------------



inline void generateRandArray(u_int numElements, float* h_input, float* max) {
  // Initialize the host input vectors
  int a;
  int b;

  for (int i = 0; i < numElements; ++i) {
    a = rand();
    b = rand();

    h_input[i] = a * 100.0 + b;

    if (h_input[i] > *max) {
      *max = h_input[i];
    }

  }
}



//-------------------



__host__ __forceinline__ void checkProcessFailure() {
  hipError_t err = hipSuccess;            // Check return values for CUDA calls

  err = hipGetLastError();
  if ( CHECK(err != hipSuccess, "Failed to launch reduceMax_persist kernel (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
}



__host__ __forceinline__ void checkResultFailure(float max, float h_max) {
  if ( max != h_max ) {
    fprintf(stderr, "Result verification failed!\n");
    fprintf(stderr, "Max should be: %f\nBut is: %f\n", max, h_max);
    exit(EXIT_FAILURE);
  } else { printf("Max value: %.6f\n", h_max); }
}



__host__ __forceinline__ void getDeviceMax(float* h_max, float* d_max) {
  hipError_t err = hipSuccess;            // Check return values for CUDA calls

  err = hipMemcpy(h_max, d_max, sizeof(u_int), hipMemcpyDeviceToHost);
  if ( CHECK(err != hipSuccess, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
}



__host__ __forceinline__ void copyHostToDeviceVector(float* d_input, float* h_input, size_t size) {
  hipError_t err = hipSuccess;            // Check return values for CUDA calls

  err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
  if ( CHECK(err != hipSuccess, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
}



//-------------------


__host__ __forceinline__ void getInput(int argc, char **argv, u_int* numElements, u_int* nR) {
  if (argc >= 2) {
    *numElements = atoi(argv[1]);
  } else {
    printf("AVISO: sem parametro de tamanho, default: 1.000\n\n");
    *numElements = 1000;
  }

  if (argc >= 3) {
    *nR = atoi(argv[2]);
  } else {
    printf("AVISO: sem parametro de repeticao, default: 30\n\n");
    *nR = 30;
  }
}


//=========================================================================



int main(int argc, char **argv) {
  hipError_t err = hipSuccess;            // Check return values for CUDA calls
  float *h_input = NULL, *d_input = NULL;   // Host and device vectors
  float h_max, *d_max;                      // Host and device max
  float max = 0;                            // max value

  u_int numElements, nR;
  getInput(argc, argv, &numElements, &nR);  // obtem inputs

  chronometer_t chrono_Normal;                     // Chronometer
  chronometer_t chrono_Atomic;                     // Chronometer
  chronometer_t chrono_Thrust;                     // Chronometer

  printf("Running reduceMax for %d elements\n", numElements);
  size_t size = numElements * sizeof(float);
  
  //------------------------ INICIA VARIAVEIS LOCAIS
  
  // Allocate the host input vector A and check
  h_input = (float *)malloc(size);
  if ( CHECK(h_input == NULL, "Failed to allocate host vectors!\n") )
    exit(EXIT_FAILURE);

  // Initialize the host input vectors
  generateRandArray(numElements, h_input, &max);

  // Reinicia os chronos
  chrono_reset(&chrono_Normal);
  chrono_reset(&chrono_Atomic);
  chrono_reset(&chrono_Thrust);

  //------------------------ COPIA DADOS PRA GPU

  // Allocate the device input vector A
  d_input = NULL;
  err = hipMalloc((void **)&d_input, size);
  if ( CHECK(err != hipSuccess, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
  d_max = NULL;
  err = hipMalloc((void **)&d_max, sizeof(u_int));
  if ( CHECK(err != hipSuccess, "Failed to allocate device max (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);

  // Copy the host input vectors A and B in host memory to the device input vectors in device memory
  //copyHostToDeviceVector(d_input, h_input, size);
  err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
  if ( CHECK(err != hipSuccess, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);

  // Initialize thrust variables
  thrust::device_ptr<float> thrust_d_ptr(d_input);
  thrust::device_vector<float> thrust_d_input(thrust_d_ptr, thrust_d_ptr+numElements);

  printf("Launching CUDA kernels with %d blocks of %d threads\n", NP*BLOCKS, THREADS);

  // EXECUTE PERSIST ============================

  //printf("\n === EXECUTANDO KERNEL PERSIST ===\n");
  //
  //for (int i = 0; i < nR; ++i) {
  //  chrono_start(&chrono_Normal);
  //
  //  reduceMax_persist<<<NP*THREADS, THREADS>>>(d_max, d_input, numElements);
  //
  //  hipDeviceSynchronize();
  //  chrono_stop(&chrono_Normal);
  //
  //  copyHostToDeviceVector(d_input, h_input, size);   // reinicia o vetor que foi alterado
  //}
  //
  //// check for error
  //checkProcessFailure();
  //
  //// Copy device max to host max
  //getDeviceMax(&h_max, d_max);
  //
  //// Verify that the result is correct
  //checkResultFailure(max, h_max);


  // EXECUTE ATOMIC =============================

  printf("\n === EXECUTANDO KERNEL ATOMIC ===\n");

  for (int i = 0; i < nR; ++i) {
    chrono_start(&chrono_Atomic);

    reduceMax_persist<<<NP*THREADS, THREADS>>>(d_max, d_input, numElements);

    hipDeviceSynchronize();
    chrono_stop(&chrono_Atomic);

    //copyHostToDeviceVector(d_input, h_input, size);   // reinicia o vetor que será alterado
  }

  // check for error
  checkProcessFailure();

  // Copy device max to host max
  getDeviceMax(&h_max, d_max);

  // Verify that the result is correct
  checkResultFailure(max, h_max);


  // EXECUTE THRUST =============================


  printf("\n === EXECUTANDO KERNEL THRUST ===\n");

  for (int i = 0; i < nR; ++i) {
    chrono_start( &chrono_Thrust );

    h_max = *(thrust::max_element(thrust_d_input.begin(), thrust_d_input.end()));

    hipDeviceSynchronize();
    chrono_stop( &chrono_Thrust );
  }

  // Verify that the result is correct
  checkResultFailure(max, h_max);


  // IMPRIME RESULTADOS ===================================


  printf("\n === RESULTADOS ===\n");

  //--

  printf("\n----THRUST\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Thrust, (char *)"thrust max_element", nR);

  double thrust_time_seconds = (double) chrono_gettotal( &chrono_Thrust )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", thrust_time_seconds );
  printf( "Vazão: %lf INT/s\n", (numElements)/thrust_time_seconds );

  //--

  printf("\n----PERSIST\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Normal, (char *)"reduceMax_persist", nR);

  double reduce_time_seconds = (double) chrono_gettotal( &chrono_Normal )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", reduce_time_seconds );
  printf( "Vazão: %lf INT/s\n", (numElements)/reduce_time_seconds );

  printf("--Tempo em relacao ao Thrust");
  printf("Em segundos: %lf", reduce_time_seconds - thrust_time_seconds);
  printf("Em porcento: %lf", (thrust_time_seconds/reduce_time_seconds)*100.0);

  //--

  printf("\n----ATOMIC\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Atomic, (char *)"reduceMax_atomic_persist", nR);

  double atomic_time_seconds = (double) chrono_gettotal( &chrono_Atomic )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", atomic_time_seconds );
  printf( "Vazão: %lf INT/s\n", (numElements)/atomic_time_seconds );

  printf("--Tempo em relacao ao Thrust");
  printf("Em segundos: %lf", atomic_time_seconds - thrust_time_seconds);
  printf("Em porcento: %lf", (thrust_time_seconds/atomic_time_seconds)*100.0);


  // FINALIZA ===================================


  // Free device and host memory
  err = hipFree(d_input);
  if ( CHECK(err != hipSuccess, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
  err = hipFree(d_max);
  if ( CHECK(err != hipSuccess, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err)) )
    exit(EXIT_FAILURE);
  free(h_input);

  printf("\nTEST FINISHED GRACIOUSLY\n");

  return 0;
}

