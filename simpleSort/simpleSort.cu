#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#define NP 28            // Number of processors
#define BLOCKS 2         // Number of blocks per processor
#define THREADS 1024     // Number of threads per block

__global__ void blocksHistoAndScan(unsigned int *HH, unsigned int *PS, int h, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
  extern __shared__ unsigned int sharedMemory[];
  unsigned int *sharedHisto = sharedMemory;
  unsigned int *sharedScan = sharedMemory + h;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = threadIdx.x; i < h; i += blockDim.x) {
    sharedHisto[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < nElements; i += stride) {
    int bin = (Input[i] - nMin) * h / (nMax - nMin + 1);
    atomicAdd(&sharedHisto[bin], 1);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < h; ++i) {
      HH[blockIdx.x * h + i] = sharedHisto[i];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    sharedScan[0] = 0;
    for (int i = 1; i < h; ++i) {
      sharedScan[i] = sharedScan[i - 1] + sharedHisto[i - 1];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < h; ++i) {
      PS[blockIdx.x * h + i] = sharedScan[i];
    }
  }
}


__global__ void globalHistoAndScan(unsigned int *HH, unsigned int *H, unsigned int *PS, unsigned int *P, int h, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
  extern __shared__ unsigned int sharedMemory[];
  unsigned int *sharedHisto = sharedMemory;
  unsigned int *sharedScan = sharedMemory + h;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = threadIdx.x; i < h; i += blockDim.x) {
    sharedHisto[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < nElements; i += stride) {
    int bin = (Input[i] - nMin) * h / (nMax - nMin + 1);
    atomicAdd(&sharedHisto[bin], 1);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < h; ++i) {
      atomicAdd(&H[i], sharedHisto[i]);
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    sharedScan[0] = 0;
    for (int i = 1; i < h; ++i) {
      sharedScan[i] = sharedScan[i - 1] + sharedHisto[i - 1];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < h; ++i) {
      atomicAdd(&P[i], sharedScan[i]);
    }
  }
}


__global__ void Partition_kernel(unsigned int *HH, unsigned int *H, unsigned int *PS, unsigned int *P, int h, unsigned int *Output, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
  extern __shared__ unsigned int sharedMemory[];
  unsigned int *sharedHisto = sharedMemory;
  unsigned int *sharedScan = sharedMemory + h;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = threadIdx.x; i < h; i += blockDim.x) {
    sharedHisto[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < nElements; i += stride) {
    int bin = (Input[i] - nMin) * h / (nMax - nMin + 1);
    int pos = P[bin] + atomicAdd(&sharedHisto[bin], 1);
    Output[pos] = Input[i];
  }
}



void verifySort(unsigned int *Input, unsigned int *Output, int nElements) {
  thrust::device_vector<unsigned int> d_Input(Input, Input + nElements);
  thrust::device_vector<unsigned int> d_Output(Output, Output + nElements);
  thrust::sort(d_Input.begin(), d_Input.end());

  bool isSorted = thrust::equal(d_Input.begin(), d_Input.end(), d_Output.begin());

  if (isSorted) { std::cout << "Sort verification: SUCCESS" << std::endl; } 
  else          { std::cout << "Sort verification: FAILURE" << std::endl; }
}



int main(int argc, char* argv[]) {
  if (argc != 4) {
    std::cerr << "Usage: ./simpleSort <nTotalElements> <h> <nR>" << std::endl;
    return 1;
  }

  std::srand(std::time(nullptr));

  int nTotalElements = std::stoi(argv[1]);                    // Numero de elementos
  int h = std::stoi(argv[2]);                                 // Numero de histogramas
  int nR = std::stoi(argv[3]);                                // Numero de chamadas do kernel
  unsigned int *Input = new unsigned int[nTotalElements];     // Vetor de entrada
  unsigned int *Output = new unsigned int[nTotalElements];    // Vetor ordenado

  // Preenche vetor
  for (int i = 0; i < nTotalElements; ++i) {
    int a = std::rand();
    int b = std::rand();
    unsigned int v = a * 100 + b;
    Input[i] = v;
  }

  // Busca menor valor e maior valor com thrust
  unsigned int nMin = *std::min_element(Input, Input + nTotalElements);
  unsigned int nMax = *std::max_element(Input, Input + nTotalElements);

  // Alocacores da GPU
  unsigned int *d_Input, *d_Output, *HH, *PS, *H, *P;
  hipMalloc((void**)&d_Input, nTotalElements * sizeof(unsigned int));
  hipMalloc((void**)&d_Output, nTotalElements * sizeof(unsigned int));
  hipMalloc((void**)&HH, 2 * h * sizeof(unsigned int)); // assuming NP=1 for nb=NP*2
  hipMalloc((void**)&PS, 2 * h * sizeof(unsigned int));
  hipMalloc((void**)&H, h * sizeof(unsigned int));
  hipMalloc((void**)&P, h * sizeof(unsigned int));

  // Copia para memoria global
  hipMemcpy(d_Input, Input, nTotalElements * sizeof(unsigned int), hipMemcpyHostToDevice);

  //for (int i = 0; i < nR; ++i) {

    blocksHistoAndScan<<<NP*BLOCKS, THREADS>>>(HH, PS, h, d_Input, nTotalElements, nMin, nMax);
    //globalHistoAndScan<<<NP*BLOCKS, THREADS>>>(HH, H, PS, P, h, d_Input, nTotalElements, nMin, nMax);
    //Partition_kernel<<<NP*BLOCKS, THREADS>>>(HH, H, PS, P, h, d_Output, d_Input, nTotalElements, nMin, nMax);

  //}

  hipMemcpy(Output, d_Output, nTotalElements * sizeof(unsigned int), hipMemcpyDeviceToHost);
  verifySort(Input, Output, nTotalElements);

  hipFree(d_Input);
  hipFree(d_Output);
  hipFree(HH);
  hipFree(PS);
  hipFree(H);
  hipFree(P);

  delete[] Input;
  delete[] Output;

  return 0;
}

