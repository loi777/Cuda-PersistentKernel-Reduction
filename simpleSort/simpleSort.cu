#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/types.h>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#define NP 28            // Number of processors
#define BLOCKS 2         // Number of blocks per processor
#define THREADS 1024     // Number of threads per block

__global__ void blocksHistoAndScan(unsigned int *HH, unsigned int *PS, int h, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
  extern __shared__ unsigned int sharedMemory[];
  unsigned int *sharedHisto = sharedMemory;
  //unsigned int *sharedScan = sharedMemory + h;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  // Vetor de histograma
  for (int i = threadIdx.x; i < h; i += blockDim.x) {
    sharedHisto[i] = 1;
  } __syncthreads();

  //for (int i = tid; i < nElements; i += stride) {
  //  int bin = (Input[i] - nMin) / (((nMax - nMin)/h) + 1);
  //  atomicAdd(&sharedHisto[bin], 1);
  //} __syncthreads();

  // Adiciona vetor a matrix de histogramas
  if (threadIdx.x == 0) {
    for (int i = 0; i < h; ++i) {
      HH[blockIdx.x * h + i] = sharedHisto[i];
    }
  } __syncthreads();

  //if (threadIdx.x == 0) {
  //  sharedScan[0] = 0;
  //  for (int i = 1; i < h; ++i) {
  //    sharedScan[i] = sharedScan[i - 1] + sharedHisto[i - 1];
  //  }
  //}
  //__syncthreads();

  //if (threadIdx.x == 0) {
  //  for (int i = 0; i < h; ++i) {
  //    PS[blockIdx.x * h + i] = sharedScan[i];
  //  }
  //}
}


//__global__ void globalHistoAndScan(unsigned int *HH, unsigned int *H, unsigned int *PS, unsigned int *P, int h, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
//  extern __shared__ unsigned int sharedMemory[];
//  unsigned int *sharedHisto = sharedMemory;
//  unsigned int *sharedScan = sharedMemory + h;
//
//  int tid = threadIdx.x + blockIdx.x * blockDim.x;
//  int stride = blockDim.x * gridDim.x;
//
//  for (int i = threadIdx.x; i < h; i += blockDim.x) {
//    sharedHisto[i] = 0;
//  }
//  __syncthreads();
//
//  for (int i = tid; i < nElements; i += stride) {
//    int bin = (Input[i] - nMin) * h / (nMax - nMin + 1);
//    atomicAdd(&sharedHisto[bin], 1);
//  }
//  __syncthreads();
//
//  if (threadIdx.x == 0) {
//    for (int i = 0; i < h; ++i) {
//      atomicAdd(&H[i], sharedHisto[i]);
//    }
//  }
//  __syncthreads();
//
//  if (threadIdx.x == 0) {
//    sharedScan[0] = 0;
//    for (int i = 1; i < h; ++i) {
//      sharedScan[i] = sharedScan[i - 1] + sharedHisto[i - 1];
//    }
//  }
//  __syncthreads();
//
//  if (threadIdx.x == 0) {
//    for (int i = 0; i < h; ++i) {
//      atomicAdd(&P[i], sharedScan[i]);
//    }
//  }
//}


//__global__ void Partition_kernel(unsigned int *HH, unsigned int *H, unsigned int *PS, unsigned int *P, int h, unsigned int *Output, unsigned int *Input, int nElements, unsigned int nMin, unsigned int nMax) {
//  extern __shared__ unsigned int sharedMemory[];
//  unsigned int *sharedHisto = sharedMemory;
//  unsigned int *sharedScan = sharedMemory + h;
//
//  int tid = threadIdx.x + blockIdx.x * blockDim.x;
//  int stride = blockDim.x * gridDim.x;
//
//  for (int i = threadIdx.x; i < h; i += blockDim.x) {
//    sharedHisto[i] = 0;
//  }
//  __syncthreads();
//
//  for (int i = tid; i < nElements; i += stride) {
//    int bin = (Input[i] - nMin) * h / (nMax - nMin + 1);
//    int pos = P[bin] + atomicAdd(&sharedHisto[bin], 1);
//    Output[pos] = Input[i];
//  }
//}


void verifySort(unsigned int *Input, unsigned int *Output, int nElements) {
  thrust::device_vector<unsigned int> d_Input(Input, Input + nElements);
  thrust::device_vector<unsigned int> d_Output(Output, Output + nElements);
  thrust::sort(d_Input.begin(), d_Input.end());

  bool isSorted = thrust::equal(d_Input.begin(), d_Input.end(), d_Output.begin());

  if (isSorted) { std::cout << "Sort verification: SUCCESS" << std::endl; } 
  else          { std::cout << "Sort verification: FAILURE" << std::endl; }
}


int main(int argc, char* argv[]) {
  if (argc != 4) {
    std::cerr << "Usage: ./simpleSort <nTotalElements> <h> <nR>" << std::endl;
    return EXIT_FAILURE;
  }

  std::srand(std::time(nullptr));
  //int nTotalElements = std::stoi(argv[1]);                    // Numero de elementos
  //int h = std::stoi(argv[2]);                                 // Numero de histogramas
  int nTotalElements = 18;
  int h = 6;
  int nR = std::stoi(argv[3]);                                // Numero de chamadas do kernel
  //unsigned int *Input = new unsigned int[nTotalElements];     // Vetor de entrada
  unsigned int Input[] = {2, 4, 33, 27, 8, 10, 42, 3, 12, 21, 10, 12, 15, 27, 38, 45, 18, 22};
  unsigned int *Output = new unsigned int[nTotalElements];    // Vetor ordenado
  unsigned int *stage = new unsigned int[nTotalElements];     // Vetor de debug da memoria da gpu

  //// Preenche vetor
  //for (int i = 0; i < nTotalElements; ++i) {
  //  int a = std::rand() % 50;
  //  int b = std::rand();
  //  unsigned int v = a * 100 + b;
  //  Input[i] = v;
  //}

  // Busca menor valor e maior valor com thrust
  unsigned int nMin = *std::min_element(Input, Input + nTotalElements);
  unsigned int nMax = *std::max_element(Input, Input + nTotalElements);

  // Alocacores da GPU
  unsigned int *d_Input, *d_Output, *HH, *PS, *H, *P;
  hipMalloc((void**)&d_Input,  nTotalElements * sizeof(unsigned int));
  hipMalloc((void**)&d_Output, nTotalElements * sizeof(unsigned int));
  hipMalloc((void**)&HH,       nTotalElements * sizeof(unsigned int)); // assuming NP=1 for nb=NP*2
  hipMalloc((void**)&PS,       nTotalElements * sizeof(unsigned int));
  hipMalloc((void**)&H,        h * sizeof(unsigned int));
  hipMalloc((void**)&P,        h * sizeof(unsigned int));

  // Copia para memoria global
  hipMemcpy(d_Input, Input, nTotalElements * sizeof(unsigned int), hipMemcpyHostToDevice);

  std::cout << "Vetor: ";
  for (size_t i=0; i<nTotalElements ;i++)
    std::cout << Input[i] << " ";
  std::cout << std::endl;
  std::cout << "nMin: " << nMin << std::endl;
  std::cout << "nMax: " << nMax << std::endl;

  //for (int i = 0; i < nR; ++i) {
    blocksHistoAndScan<<<NP*BLOCKS, THREADS>>>(HH, PS, h, d_Input, nTotalElements, nMin, nMax);
    //globalHistoAndScan<<<NP*BLOCKS, THREADS>>>(HH, H, PS, P, h, d_Input, nTotalElements, nMin, nMax);
    //Partition_kernel<<<NP*BLOCKS, THREADS>>>(HH, H, PS, P, h, d_Output, d_Input, nTotalElements, nMin, nMax);
  //}

  hipMemcpy(stage, HH, nTotalElements * sizeof(unsigned int), hipMemcpyDeviceToHost);

  std::cout << "HH: ";
  for (size_t i=0; i<nTotalElements ;i++)
    std::cout << stage[i] << " ";
  std::cout << std::endl;

  //hipMemcpy(Output, d_Output, nTotalElements * sizeof(unsigned int), hipMemcpyDeviceToHost);
  //verifySort(Input, Output, nTotalElements);

  hipFree(d_Input);
  hipFree(d_Output);
  hipFree(HH);
  hipFree(PS);
  hipFree(H);
  hipFree(P);

  //delete[] Input;
  //delete[] Output;

  return EXIT_SUCCESS;
}

