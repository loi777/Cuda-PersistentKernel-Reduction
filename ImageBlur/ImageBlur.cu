#include "hip/hip_runtime.h"
// v0.2 modified by WZ

#include <stdio.h>

//#include <wb.h>
#include "wb4.h" // use our lib instead (under construction)

typedef unsigned int u_int;

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5

//@@ INSERT CODE HERE
//@@ INSERIR AQUI o codigo do seu kernel CUDA


__global__ void rgb2uintKernelSHM(unsigned int *argb, unsigned int *rgb, int width, int height){


}


__global__ void blurKernelSHM( unsigned int *argb_out, unsigned int *argb_in, int width, int height){


}


__global__ void uint2rgbKernelSHM(unsigned int *argb, unsigned int *rgb, int width, int height){


}



int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);
  printf( "imagem de entrada: %s\n", inputImageFile );

  //  inputImage = wbImportImage(inputImageFile);
  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  // NOW: input and output images are RGB (3 channel)
  outputImage = wbImage_new(imageWidth, imageHeight, 3);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&deviceOutputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");

  //@@ INSERT CODE HERE
  //@@ INSERIR AQUI SEU codigo para ativar SEU kernel CUDA



  rgb2uintKernelSHM<<<GRID1, NT1>>>(unsigned int *argb, unsigned int *rgb, int width, int height);


  blurKernelSHM<<<yourGrid, yourBlocks>>>( unsigned int *argb_out, unsigned int *argb_in, int width, int height);


  uint2rgbKernelSHM<<<GRID1, NT1>>>(unsigned int *argb, unsigned int *rgb, int width, int height);



  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * sizeof(unsigned char) * 3, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  // DEBUG: if you want to see your image, 
  //   will generate file bellow in current directory
  wbExport( "blurred.ppm", outputImage );

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
